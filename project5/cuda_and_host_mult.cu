/**
 * In this program, we test massive parallel threading via cuda, and compare it to 
 * the single-threaded host only code. We see that the massive parallelization for matrix multiplication
 * gives us an immense boost in compute time on the GPU as opposed to host only.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


const int DSIZE = 512 * 2;
const int block_size = 16;  // CUDA maximum is 1024 total threads in block
const float A_val = 1.0f;
const float B_val = 2.0f;

// matrix multiply kernel: C = A * B
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create thread x index
  int idy = threadIdx.y+blockDim.y*blockIdx.y; // create thread y index

  if ((idx < ds) && (idy < ds)){
    float temp = 0;
    for (int i = 0; i < ds; i++)
      temp += A[idx*ds+i] * B[i*ds+idy];   // dot product of row and column
    C[idy*ds+idx] = temp;
  }
}

int main(){

    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // these are just for timing
    clock_t t0, t1, t2;
    double t1sum=0.0;
    double t2sum=0.0;

    // start timing
    t0 = clock();

    h_A = new float[DSIZE*DSIZE];
    h_B = new float[DSIZE*DSIZE];
    h_C = new float[DSIZE*DSIZE];
    for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    // Allocate device memory and copy input data over to GPU
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
    hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);

    // Cuda processing sequence step 1 is complete

    // Launch kernel
    dim3 block(block_size, block_size);  // dim3 variable holds 3 dimensions
    dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);
    mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);

    // Cuda processing sequence step 2 is complete

    // Copy results back to host
    hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);

    // Cuda processing sequence step 3 is complete

    // Verify results, and compare with the host only results.
    for (int i = 0; i < DSIZE*DSIZE; i++) if (h_C[i] != A_val*B_val*DSIZE) {
        printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], A_val*B_val*DSIZE); return -1;
    }
    printf("Success!\n"); 

    clock_t t3 = clock();
    for (int i = 0; i < DSIZE; i++) {
        for (int j = 0; j < DSIZE; j++) {
            for (int k = 0; k < DSIZE; k++) {
                h_C[i * j] += h_A[i * k] * h_B[j * k];
            }
        }
    }
    clock_t t4 = clock();
    double t4sum = ((double)(t4-t3))/CLOCKS_PER_SEC;
    printf ("Done. CPU Compute took %f seconds\n", t4sum);
    return 0;
}
  
